#include "hip/hip_runtime.h"
#include "limits.h"
#include "math.h"
#include "stdio.h"
#include "stdlib.h"

// Definitions
#define MAX_RAND 2
#define DEFAULT_NUM_ELEMENTS 16777216
#define BLOCK_SIZE 512
#define DOUBLE_BLOCK 1024
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(x) ((x) >> LOG_NUM_BANKS)

__global__ void prescanKernel(float* d_in, float* d_out, float* d_sums,
                              unsigned int N) {
  extern __shared__ float s_data[];
  unsigned int shared_end =
      DOUBLE_BLOCK + CONFLICT_FREE_OFFSET(DOUBLE_BLOCK) - 2;

  // Indexing
  unsigned int offset = 1;
  unsigned int ai = threadIdx.x;
  unsigned int bi = threadIdx.x + BLOCK_SIZE;
  unsigned int ag = ai + DOUBLE_BLOCK * blockIdx.x;
  unsigned int bg = bi + DOUBLE_BLOCK * blockIdx.x;
  unsigned int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
  unsigned int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

  // Load data into shared memory
  s_data[ai + bankOffsetA] = (ag < N) ? d_in[ag] : 0.f;
  s_data[bi + bankOffsetB] = (bg < N) ? d_in[bg] : 0.f;

  // Build sum in place up the tree
  for (unsigned int d = BLOCK_SIZE; d > 0; d >>= 1) {
    __syncthreads();
    if (threadIdx.x < d) {
      unsigned int ai = offset * (2 * threadIdx.x + 1) - 1;
      unsigned int bi = offset * (2 * threadIdx.x + 2) - 1;
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      s_data[bi] += s_data[ai];
    }
    offset <<= 1;
  }

  // Write the last element of shared memory to the auxilary array and clear it
  if (threadIdx.x == 0) {
    d_sums[blockIdx.x] = s_data[shared_end];
    s_data[shared_end] = 0.f;
  }

  // Traverse down the tree and build scan
  for (unsigned int d = 1; d <= BLOCK_SIZE; d <<= 1) {
    offset >>= 1;
    __syncthreads();
    if (threadIdx.x < d) {
      unsigned int ai = offset * (2 * threadIdx.x + 1) - 1;
      unsigned int bi = offset * (2 * threadIdx.x + 2) - 1;
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      float temp = s_data[ai];
      s_data[ai] = s_data[bi];
      s_data[bi] += temp;
    }
  }

  // Write results to global memory
  __syncthreads();
  if (ag < N) d_out[ag] = s_data[ai + bankOffsetA];
  if (bg < N) d_out[bg] = s_data[bi + bankOffsetB];
}

__global__ void additionKernel(float* d_in, float* d_out, float* d_sums,
                               unsigned int N) {
  unsigned int i = DOUBLE_BLOCK * blockIdx.x + threadIdx.x;
  if (i >= N) return;
  d_out[i] = d_in[i] + d_sums[blockIdx.x];
  d_out[i + BLOCK_SIZE] = d_in[i + BLOCK_SIZE] + d_sums[blockIdx.x];
}

void prescanOnDevice(float* h_in, float* h_out, float** d_arr, unsigned int N,
                     int tree_depth, unsigned int* lengths, dim3* dimBlock,
                     dim3* dimGrid, unsigned int shared_size, float& dur_ex,
                     float& dur_in) {
  // Setup timing
  hipEvent_t start_ex, end_ex, start_in, end_in;
  hipEventCreate(&start_ex);
  hipEventCreate(&end_ex);
  hipEventCreate(&start_in);
  hipEventCreate(&end_in);

  // Copy host array to device
  hipEventRecord(start_in, 0);
  hipMemcpy(d_arr[0], h_in, lengths[0] * sizeof(float),
             hipMemcpyHostToDevice);

  // Perform prescan on device
  hipEventRecord(start_ex, 0);
  for (int i = 0; i < tree_depth; i++)
    prescanKernel <<<dimGrid[i], dimBlock[i], shared_size>>>
        (d_arr[i], d_arr[i], d_arr[i + 1], lengths[i]);
  for (int i = tree_depth - 2; i >= 0; i--)
    additionKernel <<<dimGrid[i], dimBlock[i]>>>
        (d_arr[i], d_arr[i], d_arr[i + 1], lengths[i]);
  hipEventRecord(end_ex, 0);
  hipEventSynchronize(end_ex);

  // Copy device array back to host
  hipMemcpy(h_out, d_arr[0], lengths[0] * sizeof(float),
             hipMemcpyDeviceToHost);
  hipEventRecord(end_in, 0);
  hipEventSynchronize(end_in);

  // Calculate durations
  hipEventElapsedTime(&dur_ex, start_ex, end_ex);
  hipEventElapsedTime(&dur_in, start_in, end_in);

  // Cleanup timing
  hipEventDestroy(start_ex);
  hipEventDestroy(end_ex);
  hipEventDestroy(start_in);
  hipEventDestroy(end_in);
}

void prescanOnHost(float* h_in, float* h_ref, unsigned int N, float& dur_cpu) {
  // Setup timing
  hipEvent_t start_cpu, end_cpu;
  hipEventCreate(&start_cpu);
  hipEventCreate(&end_cpu);

  // Perform prescan on host
  hipEventRecord(start_cpu, 0);
  h_ref[0] = 0;
  for (unsigned int i = 1; i < N; i++) h_ref[i] = h_in[i - 1] + h_ref[i - 1];
  hipEventRecord(end_cpu, 0);
  hipEventSynchronize(end_cpu);

  // Calculate duration
  hipEventElapsedTime(&dur_cpu, start_cpu, end_cpu);

  // Cleanup timing
  hipEventDestroy(start_cpu);
  hipEventDestroy(end_cpu);
}

unsigned int checkResults(float* h_out, float* h_ref, unsigned int N,
                          float eps) {
  unsigned int nDiffs = 0;
  for (unsigned int i = 0; i < N; i++) {
    float delta = abs(h_out[i] - h_ref[i]);
    if (delta > eps) nDiffs++;
  }
  return nDiffs;
}

float* allocateHostArray(unsigned int size) {
  float* h_array;
  hipError_t code = hipHostMalloc(&h_array, size, hipHostMallocDefault);
  if (code != hipSuccess) {
    printf("Memory allocation on the host was unsuccessful.\n");
    exit(EXIT_FAILURE);
  }
  return h_array;
}

float* allocateDeviceArray(unsigned int size) {
  float* d_arr;
  hipError_t code = hipMalloc(&d_arr, size);
  if (code != hipSuccess) {
    printf("Memory allocation on the device was unsuccessful.\n");
    exit(EXIT_FAILURE);
  }
  return d_arr;
}

void exitUsage() {
  printf("Usage: ./p2 N [dur_max]\n");
  exit(EXIT_SUCCESS);
}

void parseInput(int argc, char** argv, unsigned int& N, float& dur_max) {
  if (argc == 1) {
    N = DEFAULT_NUM_ELEMENTS;
    dur_max = 1000.f;
    return;
  }
  if (argc != 2 && argc != 3) exitUsage();
  if (sscanf(argv[1], "%u", &N) != 1) exitUsage();
  if (argc == 2) {
    dur_max = 1000.f;
    return;
  }
  if (sscanf(argv[2], "%f", &dur_max) != 1) exitUsage();
  dur_max *= 1000;
}

int main(int argc, char** argv) {
  unsigned int N;
  float dur_max;
  parseInput(argc, argv, N, dur_max);

  // Setup timing
  int nruns_gpu = 0;
  int nruns_cpu = 0;
  float dur_ex, dur_in, dur_cpu;
  float dur_ex_total = 0.f;
  float dur_in_total = 0.f;
  float dur_cpu_total = 0.f;
  float dur_ex_min = 1e99;
  float dur_in_min = 1e99;
  float dur_cpu_min = 1e99;

  // Calculate the tree depth
  int tree_depth = 0;
  {
    unsigned int length = N;
    while (length > 1) {
      length = (length + DOUBLE_BLOCK - 1) / DOUBLE_BLOCK;
      tree_depth++;
    }
  }

  // Calculate the lengths of the device arrays
  unsigned int lengths[tree_depth + 1];
  lengths[0] = N;
  for (int i = 1; i < tree_depth + 1; i++)
    lengths[i] = (lengths[i - 1] + DOUBLE_BLOCK - 1) / DOUBLE_BLOCK;

  // Setup grid
  dim3 dimBlock[tree_depth];
  dim3 dimGrid[tree_depth];
  for (int i = 0; i < tree_depth; i++) {
    dimBlock[i].x = BLOCK_SIZE;
    dimGrid[i].x = lengths[i + 1];
  }

  // Shared memory size
  unsigned int shared_size =
      (DOUBLE_BLOCK + CONFLICT_FREE_OFFSET(DOUBLE_BLOCK)) * sizeof(float);

  // Allocate host arrays
  float* h_in = allocateHostArray(N * sizeof(float));
  float* h_out = allocateHostArray(N * sizeof(float));
  float* h_ref = allocateHostArray(N * sizeof(float));

  // Fill host array with random numbers
  srand(73);
  for (unsigned int i = 0; i < N; i++)
    // h_in[i] = ((double)rand() / RAND_MAX - 0.5f) * 2 * M;
    h_in[i] = (int)(rand() % MAX_RAND);
  // h_in[i] = 1.f;

  // Allocate device arrays
  float* d_arr[tree_depth + 1];
  for (int i = 0; i < tree_depth + 1; i++)
    d_arr[i] = allocateDeviceArray(sizeof(float) * lengths[i]);

  // Perform prescan on the device a number of times
  while (dur_in_total < dur_max) {
    nruns_gpu++;
    prescanOnDevice(h_in, h_out, d_arr, N, tree_depth, lengths, dimBlock,
                    dimGrid, shared_size, dur_ex, dur_in);
    dur_ex_total += dur_ex;
    dur_in_total += dur_in;
    if (dur_ex < dur_ex_min) dur_ex_min = dur_ex;
    if (dur_in < dur_in_min) dur_in_min = dur_in;
    if (dur_in_total == 0.f) break;
  }

  // Perform prescan on the host a number of times
  while (dur_cpu_total < dur_max) {
    nruns_cpu++;
    prescanOnHost(h_in, h_ref, N, dur_cpu);
    dur_cpu_total += dur_cpu;
    if (dur_cpu < dur_cpu_min) dur_cpu_min = dur_cpu;
    if (dur_cpu_total == 0.f) break;
  }

  dur_ex = dur_ex_total / nruns_gpu;
  dur_in = dur_in_total / nruns_gpu;
  dur_cpu = dur_cpu_total / nruns_cpu;

  // Compare device and host results
  float eps = (float)MAX_RAND * 0.001f;
  unsigned int nDiffs = checkResults(h_out, h_ref, N, eps);
  if (nDiffs == 0)
    printf("Test PASSED\n");
  else
    printf("Test FAILED; %u differences\n", nDiffs);

  // Print stuff
  printf("N: %u\n", N);
  printf("Tree depth: %d\n", tree_depth);
  printf("Block sizes: %d", dimBlock[0].x);
  for (int i = 1; i < tree_depth; i++) printf(", %d", dimBlock[i].x);
  printf("\n");
  printf("Grid sizes: %d", dimGrid[0].x);
  for (int i = 1; i < tree_depth; i++) printf(", %d", dimGrid[i].x);
  printf("\n");
  printf("GPU array lengths: %d", lengths[0]);
  for (int i = 1; i < tree_depth + 1; i++) printf(", %d", lengths[i]);
  printf("\n");
  printf("GPU last element: %24.14f\n", h_out[N - 1]);
  printf("CPU last element: %24.14f\n", h_ref[N - 1]);
  printf("Timing results %12s %12s %8s\n", "Average", "Minimum", "Num_runs");
  printf("GPU exclusive: %12.6f %12.6f %8d\n", dur_ex, dur_ex_min, nruns_gpu);
  printf("GPU inclusive: %12.6f %12.6f %8d\n", dur_in, dur_in_min, nruns_gpu);
  printf("CPU:           %12.6f %12.6f %8d\n", dur_cpu, dur_cpu_min, nruns_cpu);
  printf("\n");

  // Free arrays
  hipFree(h_in);
  hipFree(h_out);
  hipFree(h_ref);
  for (int i = 0; i < tree_depth + 1; i++) hipFree(d_arr[i]);

  return 0;
}
