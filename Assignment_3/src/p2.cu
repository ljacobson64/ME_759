#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void sumArrays(double* dA, double* dB, double* dC) {
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    dC[ind] = dA[ind] + dB[ind];
}

int int_power(int base, int exponent) {
    int result = 1;
    for (int i = 0; i < exponent; i++) { result *= base; }
    return result;
}

int main(int argc, char *argv[]) {
    if (argc != 3) {  exit(1); }
    int exponent = atoi(argv[1]);
    int nthreads = atoi(argv[2]);
    
    int N = int_power(2, exponent);  // Number of random numbers
    int nblocks = N/nthreads;
    
    // Allocate host arrays
    int bytes = sizeof(double)*N;
    double *hA, *hB, *hC, *refC, *difC;
    hA   = (double*)malloc(bytes);
    hB   = (double*)malloc(bytes);
    hC   = (double*)malloc(bytes);
    refC = (double*)malloc(bytes);
    difC = (double*)malloc(bytes);
    
    // Allocate device arrays
    double *dA, *dB, *dC;
    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);
    
    // Fill host arrays with random numbers between -10 and 10 and sum them for
    // reference
    srand(1443740650);
    for (int i = 0; i < N; i++) {
        hA[i] = ((double)rand()/(double)RAND_MAX - 0.5)*20;
        hB[i] = ((double)rand()/(double)RAND_MAX - 0.5)*20;
        refC[i] = hA[i] + hB[i];
    }
    
    // Copy host arrays to the device
    hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice);
    
    // Set up timing
    float duration_ex;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    
    // Invoke the device kernel which sums the arrays
    hipEventRecord(start, 0);
    
    sumArrays<<<nblocks, nthreads>>>(dA, dB, dC);
    
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&duration, start, end);
    hipEventDestroy(start);
    hipEventDestroy(end);
    
    // Copy the sum array back to the host
    hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost);
    
    // Calculate the difference between the sum arrays and find the maximum
    // absolute difference
    double max_dif = 0.0;
    for (int i = 0; i < N; i++) {
        difC[i] = hC[i] - refC[i];
        if (abs(difC[i]) > max_dif) { max_dif = abs(difC[i]); }
    }
    
    // Free memory
    free(hA);
    free(hB);
    free(hC);
    free(refC);
    free(difC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    // Print some information
    printf("Number of integers:    %12d\n", N);
    printf("Maximum difference:    %12.4e\n", max_dif);
    printf("Time taken:            %12.6e ms\n", duration_ex);
    
    return 0;
}
