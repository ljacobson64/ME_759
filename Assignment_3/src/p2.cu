#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void sumArrays(double* dA, double* dB, double* dC) {
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    dC[ind] = dA[ind] + dB[ind];
}

int int_power(int base, int exponent) {
    int result = 1;
    for (int i = 0; i < exponent; i++) { result *= base; }
    return result;
}

double randBetween(int low, int high) {
    double result = (double)rand()/(double)RAND_MAX*(high - low) + low;
    return result;
}

int main(int argc, char *argv[]) {
    if (argc != 3) {  exit(1); }
    
    int exponent = atoi(argv[1]);
    int N = int_power(2, exponent);  // Number of random numbers
    
    int nthreads = atoi(argv[2]);    // Number of threads per block
    int nblocks = N/nthreads;        // Number of blocks
    
    // Allocate host arrays
    int bytes = sizeof(double)*N;
    double *hA, *hB, *hC, *refC, *difC;
    hA   = (double*)malloc(bytes);
    hB   = (double*)malloc(bytes);
    hC   = (double*)malloc(bytes);
    refC = (double*)malloc(bytes);
    difC = (double*)malloc(bytes);
    
    // Allocate device arrays
    double *dA, *dB, *dC;
    hipMalloc(&dA, bytes);
    hipMalloc(&dB, bytes);
    hipMalloc(&dC, bytes);
    
    // Fill host arrays with random numbers between -10 and 10 and sum them for
    // reference
    srand(1443740650);
    for (int i = 0; i < N; i++) {
        hA[i] = randBetween(-10, 10);
        hB[i] = randBetween(-10, 10);
        refC[i] = hA[i] + hB[i];
    }
    
    // Set up timing
    struct timespec start_in, end_in;
    float duration_ex, duration_in;
    long duration_in_ns;
    hipEvent_t start_ex, end_ex;
    hipEventCreate(&start_ex);
    hipEventCreate(&end_ex);
    
    // Start inclusive timing
    clock_gettime(CLOCK_MONOTONIC, &start_in);
    
    // Copy host arrays to the device
    hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice);
    
    // Start exclusive timing
    hipEventRecord(start_ex, 0);
    
    // Invoke the device kernel which sums the arrays
    sumArrays<<<nblocks, nthreads>>>(dA, dB, dC);
    
    // End exclusive timing
    hipEventRecord(end_ex, 0);
    hipEventSynchronize(end_ex);
    
    // Copy the sum array back to the host
    hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost);
    
    // End inclusive timing
    clock_gettime(CLOCK_MONOTONIC, &end_in);
    
    // Calculate durations
    hipEventElapsedTime(&duration_ex, start_ex, end_ex);
    hipEventDestroy(start_ex);
    hipEventDestroy(end_ex);
    duration_in_ns = (end_in.tv_sec - start_in.tv_sec)*1000000000L +
                      end_in.tv_nsec - start_in.tv_nsec;
    duration_in = (float)duration_in_ns/1000000;
    
    // Calculate the difference between the sum arrays and find the maximum
    // absolute difference
    double max_dif = 0.0;
    for (int i = 0; i < N; i++) {
        difC[i] = hC[i] - refC[i];
        if (abs(difC[i]) > max_dif) { max_dif = abs(difC[i]); }
    }
    
    // Free memory
    free(hA);
    free(hB);
    free(hC);
    free(refC);
    free(difC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    
    // Print some information
    printf("Number of integers:  %12d\n", N);
    printf("Maximum difference:  %12.4e\n", max_dif);
    printf("Exclusive time:      %12.6e ms\n", duration_ex);
    printf("Inclusive time:      %12.6e ms\n", duration_in);
    
    return 0;
}
