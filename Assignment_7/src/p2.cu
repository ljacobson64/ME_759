#include "hip/hip_runtime.h"
#include "math.h"
#include "stdio.h"

#define BLOCK_SIZE 512

__global__ void reductionDevice(double* d_in, double* d_out, int N) {
  // Setup shared memory
  extern __shared__ float s_data[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // Load global memory into shared memory
  if (i < N)
    s_data[threadIdx.x] = d_in[i];
  else
    s_data[threadIdx.x] = 0.f;

  // Make sure all the memory in a block is loaded before continuing
  __syncthreads();

  // Add the first and second halves of the array and place the result in the
  // first half. Then add the first and second halves of the original first
  // half, and repeat until the final block sum is computed. The total number of
  // loops is equal to log_2(blockDim.x).
  for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
    if (threadIdx.x < offset)
      s_data[threadIdx.x] += s_data[threadIdx.x + offset];
    __syncthreads();
  }

  // Write the result for each block into d_out
  if (threadIdx.x == 0) d_out[blockIdx.x] = s_data[0];
}

void reductionHost(double* h_in, double* h_ref, int N) {
  double result = 0.f;
  for (int i = 0; i < N; i++) result += h_in[i];
  *h_ref = result;
}

bool checkResults(double* h_out, double* h_ref, double eps) {
  double delta = abs(*h_out - *h_ref);
  if (delta > eps) return false;
  return true;
}

int main(int argc, char* argv[]) {
  int N, M;
  if (argc == 3) {
    N = atoi(argv[1]);
    M = atoi(argv[2]);
  } else {
    printf("Usage: ./p2 <M> <N>\n");
    return 0;
  }
  float dur_max = 1e-30;

  // Setup timing
  int nruns_gpu = 0;
  int nruns_cpu = 0;
  float dur_ex, dur_in, dur_cpu;
  float dur_ex_total = 0.f;
  float dur_in_total = 0.f;
  float dur_cpu_total = 0.f;

  // For N = 50,000,000 and BLOCK_SIZE = 512:
  //   sizes[0] = 50,000,000
  //   sizes[1] =     97,657
  //   sizes[2] =        191
  //   sizes[3] =          1
  int sizes[4];
  sizes[0] = N;
  sizes[1] = (sizes[0] + BLOCK_SIZE - 1) / BLOCK_SIZE;
  sizes[2] = (sizes[1] + BLOCK_SIZE - 1) / BLOCK_SIZE;
  sizes[3] = (sizes[2] + BLOCK_SIZE - 1) / BLOCK_SIZE;

  int shared_size = sizeof(double) * BLOCK_SIZE;

  // Allocate host arrays
  double* h_in, *h_out, *h_ref;
  hipHostMalloc(&h_in, sizeof(double) * N, hipHostMallocDefault);
  hipHostMalloc(&h_out, sizeof(double), hipHostMallocDefault);
  hipHostMalloc(&h_ref, sizeof(double), hipHostMallocDefault);

  // Allocate device arrays
  double* d_0, *d_1, *d_2, *d_3;
  hipMalloc(&d_0, sizeof(double) * sizes[0]);
  hipMalloc(&d_1, sizeof(double) * sizes[1]);
  hipMalloc(&d_2, sizeof(double) * sizes[2]);
  hipMalloc(&d_3, sizeof(double) * sizes[3]);

  // Fill host array with random numbers
  srand(73);
  for (int i = 0; i < N; i++)
    h_in[i] = ((double)rand() / RAND_MAX - 0.5f) * 2 * M;

  while (dur_in_total < dur_max) {
    nruns_gpu++;

    // Setup timing
    hipEvent_t start_ex, end_ex, start_in, end_in;
    hipEventCreate(&start_ex);
    hipEventCreate(&end_ex);
    hipEventCreate(&start_in);
    hipEventCreate(&end_in);

    // Copy host array to device
    hipEventRecord(start_in, 0);
    hipMemcpy(d_0, h_in, N * sizeof(double), hipMemcpyHostToDevice);

    // Perform reduction on device
    hipEventRecord(start_ex, 0);
    reductionDevice <<<sizes[1], BLOCK_SIZE, shared_size>>>
        (d_0, d_1, sizes[0]);
    reductionDevice <<<sizes[2], BLOCK_SIZE, shared_size>>>
        (d_1, d_2, sizes[1]);
    reductionDevice <<<sizes[3], BLOCK_SIZE, shared_size>>>
        (d_2, d_3, sizes[2]);
    hipEventRecord(end_ex, 0);
    hipEventSynchronize(end_ex);

    // Copy device array back to host
    hipMemcpy(h_out, d_3, sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(end_in, 0);
    hipEventSynchronize(end_in);

    // Calculate durations
    hipEventElapsedTime(&dur_ex, start_ex, end_ex);
    hipEventElapsedTime(&dur_in, start_in, end_in);
    dur_ex_total += dur_ex;
    dur_in_total += dur_in;
  }

  while (dur_cpu_total < dur_max) {
    nruns_cpu++;

    // Setup timing
    hipEvent_t start_cpu, end_cpu;
    hipEventCreate(&start_cpu);
    hipEventCreate(&end_cpu);

    // Perform reduction on host
    hipEventRecord(start_cpu, 0);
    reductionHost(h_in, h_ref, N);
    hipEventRecord(end_cpu, 0);
    hipEventSynchronize(end_cpu);

    // Calculate durations
    hipEventElapsedTime(&dur_cpu, start_cpu, end_cpu);
    dur_cpu_total += dur_cpu;
  }

  dur_ex = dur_ex_total / nruns_gpu;
  dur_in = dur_in_total / nruns_gpu;
  dur_cpu = dur_cpu_total / nruns_cpu;

  // Compare device and host results
  double eps = (double)M * 2 * 0.001f;
  bool testPassed = checkResults(h_out, h_ref, eps);
  if (testPassed)
    printf("Test PASSED\n");
  else
    printf("Test FAILED\n");

  // Print stuff
  printf("GPU result: %20.14f\n", *h_out);
  printf("CPU result: %20.14f\n", *h_ref);
  printf("Num runs GPU: %10d\n", nruns_gpu);
  printf("Num runs CPU: %10d\n", nruns_cpu);
  printf("GPU execution time (exclusive): %12.6f\n", dur_ex);
  printf("GPU execution time (inclusive): %12.6f\n", dur_in);
  printf("CPU execution time:             %12.6f\n", dur_cpu);
  printf("\n");

  // Free arrays
  hipFree(h_in);
  hipFree(h_out);
  hipFree(h_ref);
  hipFree(d_0);
  hipFree(d_1);
  hipFree(d_2);
  hipFree(d_3);

  return 0;
}
