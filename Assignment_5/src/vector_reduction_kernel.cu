/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

#ifndef _VECTOR_REDUCTION_KERNEL_H_
#define _VECTOR_REDUCTION_KERNEL_H_

////////////////////////////////////////////////////////////////////////////////
//! @param gi_data  input data in global memory
//! @param go_data  output data in global memory
//! @param n        input number of elements to scan from input data
////////////////////////////////////////////////////////////////////////////////
//__global__ void reduction(float *gi_data, float *go_data, int n) {
//  // Placeholder
//}

__global__ void reduce0(float *gi_data, float *go_data, int n) {
  extern __shared__ float sdata[];
  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = gi_data[i];
  __syncthreads();
  // do reduction in shared mem
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    int index = 2 * s * tid;
    if (index < blockDim.x) {
      sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0) go_data[blockIdx.x] = sdata[0];
}

#endif  // #ifndef _VECTOR_REDUCTION_KERNEL_H_
