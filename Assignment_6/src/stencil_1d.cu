#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include "stdio.h"
#include "stdlib.h"

#define RADIUS 3
#define BLOCK_SIZE 512
#define MAX_GRID_WIDTH 49152

int checkResults(int startElem, int endElem, float *cudaRes, float *res) {
  int nDiffs = 0;
  const float smallVal = 0.000001f;
  for (int i = startElem; i < endElem; i++)
    if (fabs(cudaRes[i] - res[i]) > smallVal) nDiffs++;
  return nDiffs;
}

void initializeWeights(float *weights) {
  // Hardcoded for RADIUS = 3
  weights[0] = 0.50f;
  weights[1] = 0.75f;
  weights[2] = 1.25f;
  weights[3] = 2.00f;
  weights[4] = 1.25f;
  weights[5] = 0.75f;
  weights[6] = 0.50f;
}

void initializeArray(float *arr, int nElements) {
  const int myMinNumber = -5;
  const int myMaxNumber = 5;
  srand(time(NULL));
  for (int i = 0; i < nElements; i++)
    arr[i] = (float)(rand() % (myMaxNumber - myMinNumber + 1) + myMinNumber);
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in,
                        float *out) {
  for (int i = sIdx; i < eIdx; i++) {
    out[i] = 0.f;
    out[i] += weights[0] * in[i - RADIUS];
    out[i] += weights[1] * in[i - RADIUS + 1];
    out[i] += weights[2] * in[i - RADIUS + 2];
    out[i] += weights[3] * in[i - RADIUS + 3];
    out[i] += weights[4] * in[i - RADIUS + 4];
    out[i] += weights[5] * in[i - RADIUS + 5];
    out[i] += weights[6] * in[i - RADIUS + 6];
    out[i] /= 7.f;
  }
}

__global__ void applyStencil1D_V4(int sIdx, int eIdx, const float *weights,
                                  float *in, float *out) {
  int i = sIdx + (blockIdx.x * blockDim.x + threadIdx.x) +
          blockDim.x * gridDim.x * blockIdx.y;

  if (i >= eIdx) return;

  float result = 0.f;
  result += weights[0] * in[i - 3];
  result += weights[1] * in[i - 2];
  result += weights[2] * in[i - 1];
  result += weights[3] * in[i];
  result += weights[4] * in[i + 1];
  result += weights[5] * in[i + 2];
  result += weights[6] * in[i + 3];
  result /= 7.f;
  out[i] = result;
}

__global__ void applyStencil1D_V5(int sIdx, int eIdx, const float *weights,
                                  float *in, float *out) {
  extern __shared__ float sdata[];
  int i = sIdx + (blockIdx.x * blockDim.x + threadIdx.x) +
          blockDim.x * gridDim.x * blockIdx.y;

  if (i >= eIdx) return;

  // Read into shared memory
  sdata[threadIdx.x + RADIUS] = in[i];
  if (threadIdx.x < RADIUS) {
    sdata[threadIdx.x] = in[i - RADIUS];
    sdata[threadIdx.x + RADIUS + BLOCK_SIZE] = in[i + BLOCK_SIZE];
  }

  __syncthreads();

  // Calculate result
  float result = 0.f;
  result += weights[0] * sdata[threadIdx.x];
  result += weights[1] * sdata[threadIdx.x + 1];
  result += weights[2] * sdata[threadIdx.x + 2];
  result += weights[3] * sdata[threadIdx.x + 3];
  result += weights[4] * sdata[threadIdx.x + 4];
  result += weights[5] * sdata[threadIdx.x + 5];
  result += weights[6] * sdata[threadIdx.x + 6];
  result /= 7.f;
  out[i] = result;
}

int int_power(int x, int n) {
  if (x == 0) return 0;
  if (n <= 0) return 1;
  int y = 1;
  while (n > 1) {
    if (n % 2 == 0) {
      x *= x;
      n /= 2;
    } else {
      y *= x;
      x *= x;
      n = (n - 1) / 2;
    }
  }
  return x * y;
}

int main(int argc, char *argv[]) {
  int version;
  int N;
  if (argc == 3) {
    version = atoi(argv[1]);
    N = int_power(10, atoi(argv[2]));
  } else {
    printf("Usage: ./p1 <kernel_version> <log10(N)>\n");
    printf("Allowed versions: 4, 5, 6\n");
    return 0;
  }

  int wsize = (2 * RADIUS + 1) * sizeof(float);
  int size = N * sizeof(float);

  // Setup timing
  float dur_ex, dur_in, dur_cpu;
  float dur_ex_total = 0.f;
  float dur_in_total = 0.f;
  float dur_cpu_total = 0.f;
  float dur_max = 1000.f;
  int num_runs_gpu = 0;
  int num_runs_cpu = 0;

  // Allocate host resources
  float *weights, *in, *out, *cuda_out;
  if (version == 4 || version == 5) {
    weights = (float *)malloc(wsize);
    in = (float *)malloc(size);
    out = (float *)malloc(size);
    cuda_out = (float *)malloc(size);
  } else if (version == 6) {
    hipHostMalloc(&weights, wsize);
    hipHostMalloc(&in, size);
    hipHostMalloc(&out, size);
    hipHostMalloc(&cuda_out, size);
  }

  // Allocate device resources
  float *d_weights, *d_in, *d_out;
  hipMalloc(&d_weights, wsize);
  hipMalloc(&d_in, size);
  hipMalloc(&d_out, size);

  // Fill weights and array
  initializeWeights(weights);
  initializeArray(in, N);

  // Setup grid
  dim3 dimBlock, dimGrid;
  dimBlock.x = BLOCK_SIZE;
  int num_grids = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int shared_size = (BLOCK_SIZE + 2 * RADIUS) * sizeof(float);
  if (num_grids <= MAX_GRID_WIDTH) {
    dimGrid.x = num_grids;
    dimGrid.y = 1;
  } else {
    dimGrid.x = MAX_GRID_WIDTH;
    dimGrid.y = (num_grids + MAX_GRID_WIDTH - 1) / MAX_GRID_WIDTH;
  }

  while (dur_in_total < dur_max) {
    num_runs_gpu += 1;

    // Setup timing
    hipEvent_t start_ex, end_ex, start_in, end_in;
    hipEventCreate(&start_ex);
    hipEventCreate(&end_ex);
    hipEventCreate(&start_in);
    hipEventCreate(&end_in);

    // Start inclusive timing
    hipEventRecord(start_in, 0);

    // Copy to device
    hipMemcpy(d_weights, weights, wsize, hipMemcpyHostToDevice);
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    // Start exclusive timing
    hipEventRecord(start_ex, 0);

    // Execute kernel
    if (version == 4)
      applyStencil1D_V4 <<<dimGrid, dimBlock>>>
          (RADIUS, N - RADIUS, d_weights, d_in, d_out);
    else if (version == 5 || version == 6)
      applyStencil1D_V5 <<<dimGrid, dimBlock, shared_size>>>
          (RADIUS, N - RADIUS, d_weights, d_in, d_out);

    // End exclusive timing
    hipEventRecord(end_ex, 0);
    hipEventSynchronize(end_ex);

    // Copy from device
    hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);

    // End inclusive timing
    hipEventRecord(end_in, 0);
    hipEventSynchronize(end_in);

    // Calculate durations
    hipEventElapsedTime(&dur_ex, start_ex, end_ex);
    hipEventElapsedTime(&dur_in, start_in, end_in);

    dur_ex_total += dur_ex;
    dur_in_total += dur_in;
  }

  while (dur_cpu_total < dur_max) {
    num_runs_cpu += 1;

    // Setup timing
    hipEvent_t start_cpu, end_cpu;
    hipEventCreate(&start_cpu);
    hipEventCreate(&end_cpu);

    // Run on CPU
    hipEventRecord(start_cpu, 0);
    applyStencil1D_SEQ(RADIUS, N - RADIUS, weights, in, out);
    hipEventRecord(end_cpu, 0);
    hipEventSynchronize(end_cpu);
    hipEventElapsedTime(&dur_cpu, start_cpu, end_cpu);

    dur_cpu_total += dur_cpu;
  }

  // Compare GPU result to CPU result
  int nDiffs = checkResults(RADIUS, N - RADIUS, cuda_out, out);
  if (nDiffs == 0)
    printf("Looks good.\n");
  else
    printf("Doesn't look good: %d differences\n", nDiffs);

  // Calculate average durations
  dur_ex = dur_ex_total / num_runs_gpu;
  dur_in = dur_in_total / num_runs_gpu;
  dur_cpu = dur_cpu_total / num_runs_cpu;

  // Print stuff
  printf("Version: %u\n", version);
  printf("N: 10^%d = %lu\n", atoi(argv[2]), N);
  printf("blockDim.x: %u\n", dimBlock.x);
  printf("blockDim.y: %u\n", dimBlock.y);
  printf("gridDim.x:  %u\n", dimGrid.x);
  printf("gridDim.y:  %u\n", dimGrid.y);
  printf("Num runs GPU: %u\n", num_runs_gpu);
  printf("Num runs CPU: %u\n", num_runs_cpu);
  printf("GPU execution time (exclusive): %15.6f ms\n", dur_ex);
  printf("GPU execution time (inclusive): %15.6f ms\n", dur_in);
  printf("CPU execution time:             %15.6f ms\n", dur_cpu);
  printf("\n");

  // Free resources
  if (version == 4 || version == 5) {
    free(weights);
    free(in);
    free(out);
    free(cuda_out);
  } else if (version == 6) {
    hipFree(weights);
    hipFree(in);
    hipFree(out);
    hipFree(cuda_out);
  }
  hipFree(d_weights);
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
