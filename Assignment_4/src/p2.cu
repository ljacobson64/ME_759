#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE 16

// Kernel using global memory
__global__ void cudaMultiplyArraysGlobal(int* dA, int* dB, int* dC,
    int hA, int wA, int hB, int wB, int hC, int wC) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  
  int y = by*BLOCK_SIZE + ty;  // row
  int x = bx*BLOCK_SIZE + tx;  // column

  if (y > hA || x > wB) return;

  int result = 0;
  for (int e = 0; e < wA; e++)
    result += dA[y*wA + e]*dB[e*wB + x];

  dC[y*wC + x] = result;
}

// Kernel using shared memory
//
// Right now this works if and only if the matrices have dimensions that are
// integer multiples of the block size; i.e. (16x32)*(32,1) would not work
// because 1 % 16 != 0
__global__ void cudaMultiplyArraysShared(int* dA, int* dB, int* dC,
    int hA, int wA, int hB, int wB, int hC, int wC) {
  // Thread and block indices
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Number of subarrays for each block
  int nsubs = wA/BLOCK_SIZE;

  // Initialize subarrays in shared memory
  __shared__ int sdA[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int sdB[BLOCK_SIZE][BLOCK_SIZE];

  // Loop over each subarray
  int result = 0;
  for (int i = 0; i < nsubs; i++) {
    sdA[ty][tx] = dA[(by*BLOCK_SIZE + ty)*wA + ( i*BLOCK_SIZE + tx)];
    sdB[ty][tx] = dB[( i*BLOCK_SIZE + ty)*wB + (bx*BLOCK_SIZE + tx)];

    __syncthreads();

    for (int k = 0; k < BLOCK_SIZE; k++)
      result += sdA[ty][k]*sdB[k][tx];

    __syncthreads();
  }

  // Fill result array
  dC[(by*BLOCK_SIZE + ty)*wB + (bx*BLOCK_SIZE + tx)] = result;
}

void fill_array(int* A, int hA, int wA) {
  for (int i = 0; i < hA; i++)
    for (int j = 0; j < wA; j++)
      A[i*wA + j] = i + j;
}

int main() {
  // Array sizes
  int m = 16;
  int n = 32;
  int p =  1;
  int hA = m, wA = n;
  int hB = n, wB = p;
  int hC = m, wC = p;
  int sA = hA*wA;
  int sB = hB*wB;
  int sC = hC*wC;

  // Allocate host arrays
  int *A, *B, *C;
  A = (int*)malloc(sizeof(int)*sA);
  B = (int*)malloc(sizeof(int)*sB);
  C = (int*)malloc(sizeof(int)*sC);

  // Allocate device arrays
  int *dA, *dB, *dC;
  hipMalloc(&dA, sizeof(int)*sA);
  hipMalloc(&dB, sizeof(int)*sB);
  hipMalloc(&dC, sizeof(int)*sC);

  // Fill A and B with some integers
  fill_array(A, hA, wA);
  fill_array(B, hB, wB);

  // Set up block grid
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((wC + BLOCK_SIZE - 1)/BLOCK_SIZE,
               (hC + BLOCK_SIZE - 1)/BLOCK_SIZE);

  // Set up timing
  struct timespec start_in, end_in;
  int num_runs = 65536;
  long dur_in_ns;
  double dur_in = 0.0, dur_in_total = 0.0;
  double dur_in_max = 0.0, dur_in_min = 1e99;

  for (int i = 0; i < num_runs; i++) {
    // Start inclusive timing
    clock_gettime(CLOCK_MONOTONIC, &start_in);

    // Copy host arrays to the device
    hipMemcpy(dA, A, sizeof(int)*sA, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(int)*sB, hipMemcpyHostToDevice);

    // Invoke the device kernel which multiplies the arrays using global memory
    cudaMultiplyArraysGlobal<<<dimGrid, dimBlock>>>(dA, dB, dC,
                                                    hA, wA, hB, wB, hC, wC);

    // The kernel with shared memory does work, but only when the matrix
    // dimensions are integer multiples of the block size
    //cudaMultiplyArraysShared<<<dimGrid, dimBlock>>>(dA, dB, dC,
    //                                                hA, wA, hB, wB, hC, wC);

    // Copy the result array back to the host
    hipMemcpy(C, dC, sizeof(int)*sC, hipMemcpyDeviceToHost);

    // End inclusive timing
    clock_gettime(CLOCK_MONOTONIC, &end_in);

    // Calculate duration
    dur_in_ns = (end_in.tv_sec - start_in.tv_sec)*1000000000l +
                 end_in.tv_nsec - start_in.tv_nsec;
    dur_in = (double)(dur_in_ns/1000000.0);
    dur_in_total += dur_in;
    if (dur_in > dur_in_max) dur_in_max = dur_in;
    if (dur_in < dur_in_min) dur_in_min = dur_in;
  }

  // Write result to file
  FILE *fp;
  fp = fopen("problem2.out", "w");
  for (int i = 0; i < hC; i++) {
    for (int j = 0; j < wC; j++)
      fprintf(fp, "%12d ", C[i*wC + j]);
    fprintf(fp, "\n");
  }
  fclose(fp);

  // Free memory
  free(A);
  free(B);
  free(C);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  // Get device properties
  hipDeviceProp_t gpu_props;
  hipGetDeviceProperties(&gpu_props, 0);

  // Print some information
  printf("Device name: %s\n", gpu_props.name);
  printf("Dimension 1 (m): %12d\n", m);
  printf("Dimension 2 (n): %12d\n", n);
  printf("Dimension 3 (p): %12d\n", p);
  printf("Block size:      %12d\n", BLOCK_SIZE);
  printf("Number of runs:  %12d\n", num_runs);
  printf("Inclusive time (maximum): %12.6f ms\n", dur_in_max);
  printf("Inclusive time (average): %12.6f ms\n", dur_in_total/num_runs);
  printf("Inclusive time (minimum): %12.6f ms\n", dur_in_min);

  return 0;
}
